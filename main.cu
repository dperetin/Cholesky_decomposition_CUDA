#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
//#include <cutil.h>
#include <math.h>
#include <hdf5.h>

using namespace std;

void cpu_potrf(float *m_in, float *m_out, int size)
{
    for (int i = 0; i < size; i++) {
        float sum = 0;
        for (int k = 0; k < i; k++) {
            sum += (m_out[k * size + i] * m_out[k * size + i]);
        }
        m_out[i * size + i] = sqrt(m_in[i * size + i] - sum);
        for (int j = i + 1; j < size; j++ ) {
            sum = 0;
            for (int k = 0; k < i; k++) {
                sum += (m_out[k * size + i] * m_out[k * size + j]);
            }
            m_out[i * size + j] = (m_in[i * size + j] - sum) / 
                                   m_out[i * size + i];
        }
    }
}

void standard (float *A, float  *B, float *C, int size)
{
    int i, j, k;

    for (i = 0; i < size; i++)
        for (j = 0; j < size; j++)
            for (k = 0; k < size; k++) {
                C[i * size + j] += A[k * size + i] * B[k * size + j]; 
                if (i==j)
                    C[i*size+j]+= 0.001;
            }
}

void init(float *v, int n)
{
    int i;
    srand(time(NULL));
    for (i = 0; i < n; i++)
        v[i] = rand() / (float(RAND_MAX) + 1) - 1; 
}

void loadMatrix(float * matrix, char *s, int size)
{
    fstream f;
    int i = 0;
    f.open(s, ifstream::in);
    while (f.good()) {
        f >> matrix[i];
        i++;
    }
    f.close();
}

void saveMatrix(float * matrix, char *s, int size)
{
    fstream f;
    f.open(s, ifstream::out);
    for(int i = 0; i < size; i++) {
        for(int j = 0; j < size; j++) {
            f << matrix[i * size + j] << " ";
        }
        f << endl;
    }
    f.close();
}

__global__ void gpu_potrf(float (*m)[512], int size, int p)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float a[16][16 + 1];
    a[ty][tx] = m[(ty + 16 * p)][tx + 16 * p];

    __syncthreads();

    float d;

    #pragma unroll 16
    for (int k = 0; k < 16; k++) {
        __syncthreads();

        d = rsqrt(a[k][k]);

        __syncthreads();

        if ((ty == k) && (tx >= k)) 
            a[tx][ty] = (a[tx][ty]) * d;
    
        __syncthreads();

        if ((ty >= tx) && (tx > k)) 
            a[ty][tx] = a[ty][tx] - a[tx][k] * a[ty][k]; 
    }

    __syncthreads();

    if (ty >= tx) 
        m[(tx + 16 * p) ][ty + 16 * p] = a[ty][tx];
    
}

__global__ void gpu_inv_l(float (*u)[512], int size, int p)
{
    int i, j;

    int tid = threadIdx.x;
    int bx = blockIdx.x + 1;

    __shared__ float b[16][16];

    for(i = 0; i < 16; i++)
        b[i][tid] = u[(i + p * 16) ][ tid + (bx + p) * 16];

    b[0][tid] = b[0][tid] / u[(0 + p * 16) ][ + (0 + 16 * p)];

    for (i = 1; i < 16; i++){
        for (j = 0; j < i; j++) {
            b[i][tid] = b[i][tid] - 
                u[(j + p * 16) ][ (i + p * 16)] * b[j][tid];
        }
        b[i][tid] = b[i][tid] / u[(i + p * 16) ][ + (i + 16 *p)];
    }

    for(i = 0; i < 16; i++)
        u[(i + p * 16) ][ tid + (bx + p) * 16] = b[i][tid];
}

__global__ void gpu_mm_a(float (*m)[512], int size, int p, int s, int mod, int visina)
{
    __shared__ float s_a1[16][16];
    __shared__ float s_a2[16][16];
    __shared__ float s_b1[16][16];
    __shared__ float s_b2[16][16];
    //__shared__ float s_c[16][16];
    float s_c1 = 0, s_c2 = 0, s_c3 = 0, s_c4 = 0;
    int tx = threadIdx.x, i;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    
    if (bx + 1 != gridDim.x)
    {
      s_a1[ty][tx] = m[(ty + p * 16) ][ tx + (s) * 16];
    s_a2[ty][tx] = m[(ty + p * 16) ][ tx + (s + 1) * 16];
    s_b1[ty][tx] = m[(ty + p * 16) ][ tx + (s + bx * 2) * 16];
    s_b2[ty][tx] = m[(ty + p * 16) ][ tx + (s + bx * 2 + 1) * 16];

    __syncthreads();

    #pragma unroll 16
    for (i = 0; i < 16; i++)
    {
        s_c1 += s_a1[i][ty] * s_b1[i][tx];
        s_c2 += s_a1[i][ty] * s_b2[i][tx];
        s_c3 += s_a2[i][ty] * s_b1[i][tx];
        s_c4 += s_a2[i][ty] * s_b2[i][tx];
    }
    
    
    m[(ty + (s) * 16) ][ tx + (s + bx * 2) * 16] -= s_c1;
    m[(ty + (s) * 16) ][ tx + (s + (bx *2)+ 1) * 16] -= s_c2;
    m[(ty + (s+1) * 16) ][ tx + (s + (bx * 2)) * 16] -= s_c3;
    m[(ty + (s+1) * 16) ][ tx + (s + (bx * 2)+ 1) * 16] -= s_c4;
    return;
    }
    if (bx + 1 == gridDim.x && mod == 1 && visina == 1) {
        s_a1[ty][tx] = m[(ty + p * 16) ][ tx + (s) * 16];
        s_b1[ty][tx] = m[(ty + p * 16) ][ tx + (s + bx * 2) * 16];
        __syncthreads();
        #pragma unroll 16
        for (i = 0; i < 16; i++)
        {
            s_c1 += s_a1[i][ty] * s_b1[i][tx];
            
        }
        m[(ty + (s) * 16) ][ tx + (s + bx * 2) * 16] -= s_c1;
        return;
    }
    if (bx + 1 == gridDim.x && mod == 0) 
        return;
    if (bx + 1 == gridDim.x && mod == 1) {
        s_a1[ty][tx] = m[(ty + p * 16) ][ tx + (s) * 16];
        s_a2[ty][tx] = m[(ty + p * 16) ][ tx + (s + 1) * 16];
        s_b1[ty][tx] = m[(ty + p * 16) ][ tx + (s + bx * 2) * 16];
        __syncthreads();
        #pragma unroll 16
        for (i = 0; i < 16; i++)
        {
            s_c1 += s_a1[i][ty] * s_b1[i][tx];
            s_c3 += s_a2[i][ty] * s_b1[i][tx];
            
        }
        m[(ty + (s) * 16) ][tx + (s + bx * 2) * 16] -= s_c1;
        m[(ty + (s+1) * 16) ][tx + (s + (bx * 2)) * 16] -= s_c3;
        return;
    }
    
    
}

int main(int argc, char *argv[])
{

    if (argc < 2 || argc > 4)
    {
        fprintf(stderr, "GPUCHOL [red matrice] [opcionalno - ime datoteke]");
        return 1;
    }

    int size = atoi(argv[1]);

    hid_t       file_id, dataset_id;
    
    float *m_in, (*device_m)[512], *v, *cpu_rez;
    m_in = new float[size * size];
//  m_out = new float[size * size];
    
    memset(m_in, 0, size * size * sizeof(float));
//  memset(m_out, 0, size * size * sizeof(float));
    
    

    int deviceOrdinal = 0;
    hipSetDevice(deviceOrdinal);
    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, deviceOrdinal);
    printf("\n%s\n\n", device_properties.name);


    if (argc == 2) {

        cpu_rez = new float[size * size];
        v = new float[size * size];
        memset(cpu_rez, 0, size * size * sizeof(float));

        printf("Generiranje matrice:\t\t");
        fflush(stdout);

//      CUT_SAFE_CALL(cutCreateTimer(&t_mat_gen));
//      CUT_SAFE_CALL(cutStartTimer(t_mat_gen));
    
        
        init(v, size * size);
        standard(v, v, m_in, size);

//      CUT_SAFE_CALL(cutStopTimer(t_mat_gen));

//      printf("%f\n", cutGetTimerValue(t_mat_gen));

        printf("CPU racuna:\t\t\t");

//      CUT_SAFE_CALL(cutCreateTimer(&t_cpu));
//      CUT_SAFE_CALL(cutStartTimer(t_cpu));
    
        cpu_potrf(m_in, cpu_rez, size);
//      CUT_SAFE_CALL(cutStopTimer(t_cpu));

//      printf("%f\n\n", cutGetTimerValue(t_cpu));

//      saveMatrix(m_in, "m.mat", size);
//      saveMatrix(cpu_rez, "cpu_rez.mat", size);
    }

    if (argc == 3) {

        printf("Ucitavanje matrice iz datoteke:\t");
        fflush(stdout);

//      CUT_SAFE_CALL(cutCreateTimer(&t_mat_load));
//      CUT_SAFE_CALL(cutStartTimer(t_mat_load));
    
        file_id = H5Fopen(argv[2], H5F_ACC_RDWR, H5P_DEFAULT);
        dataset_id = H5Dopen(file_id, "/16", H5P_DEFAULT);
        H5Dread(dataset_id, H5T_IEEE_F32LE, 
                         H5S_ALL, H5S_ALL, H5P_DEFAULT, m_in);
        H5Dclose(dataset_id);
        H5Fclose(file_id);
        
//      CUT_SAFE_CALL(cutStopTimer(t_mat_load));

//      printf("%f\n\n", cutGetTimerValue(t_mat_load));

    }

    // GPU //
    int n = size;
    hipEvent_t start, stop;
    dim3 blokovaPoGridu, thredovaPoBloku;
    
    thredovaPoBloku.x = 16;
    thredovaPoBloku.y = 16;

    hipMalloc((void **) &device_m, n * n * sizeof(float));

    printf("Kopiranje matrice na GPU:\t");

//  CUT_SAFE_CALL(cutCreateTimer(&t_h2d));
//  CUT_SAFE_CALL(cutStartTimer(t_h2d));
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMemcpy(device_m, m_in, n * n * sizeof(float), hipMemcpyHostToDevice);


//  CUT_SAFE_CALL(cutStopTimer(t_h2d));

//  printf("%f\n", cutGetTimerValue(t_h2d));

    printf("GPU racuna:\t\t\t");



//  CUT_SAFE_CALL(cutCreateTimer(&t_gpu));
//  CUT_SAFE_CALL(cutStartTimer(t_gpu));

    int i, j;
    int it = n / 16 - 1;
    gpu_potrf <<<1, thredovaPoBloku>>> (device_m, size, 0);

    for (i = 0; i < n / 16 - 1; i++) {

        gpu_inv_l <<<it, 16>>> (device_m, size, i);
        
        for (j = i; j < n / 16 - 1; j += 2)
        
            gpu_mm_a <<<(n / 16 - 1 - j) / 2 + 1, thredovaPoBloku>>> 
                (device_m, size, i, j + 1, (n / 16 - 1 - j) % 2, n/16 - (j+1));
    
        gpu_potrf <<<1, thredovaPoBloku>>> (device_m, size, i + 1);
    
        it--;
    }
    


//  CUT_SAFE_CALL(cutStopTimer(t_gpu));

//  printf("%f\n", cutGetTimerValue(t_gpu));
    

    printf("Kopiranje matrice natrag:\t");
//  CUT_SAFE_CALL(cutCreateTimer(&t_d2h));
//  CUT_SAFE_CALL(cutStartTimer(t_d2h));

    hipMemcpy(m_in, device_m, 
            n * n * sizeof(float), hipMemcpyDeviceToHost);

    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float vrijeme;
    hipEventElapsedTime(&vrijeme, start, stop);
    
//  CUT_SAFE_CALL(cutStopTimer(t_d2h));

//  printf("%f\n", cutGetTimerValue(t_d2h));
//  printf("----------------------------------------------\n");
//  printf("UKUPNO:\t\t\t\t%f\n\n", cutGetTimerValue(t_d2h) + 
//                         cutGetTimerValue(t_h2d) + 
//                         cutGetTimerValue(t_gpu));
    printf("\nUKUPNO: %f\n", vrijeme);
    if (argc == 2) {
        printf("CPU %.13f\n", cpu_rez[(size - 1) * size + size - 1]);
    }
    printf("GPU %.13f\n", m_in[(size - 1) * size + size - 1]);


//  saveMatrix(m_out, "rez.mat", n);

    free(m_in);
//  free(m_out);
    if (argc == 2) {
        free(cpu_rez);
        free(v);
    }
    hipFree(device_m);

    return 0;
}

