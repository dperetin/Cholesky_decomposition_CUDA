#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <cutil.h>
#include <math.h>

using namespace std;

void cpu_dpotrf(float *m_in, float *m_out, int size)
{
	    for (int i = 0; i < size; i++) {
			        float sum = 0;
					        for (int k = 0; k < i; k++){
								            sum += (m_out[k * size + i] * m_out[k * size + i]);
											        }
							        m_out[i * size + i] = sqrt(m_in[i * size + i] - sum);
									        for (int j = i + 1; j < size; j++ ) {
												            sum = 0;
															            for (int k = 0; k < i; k++){
																			                sum += (m_out[k * size + i] * m_out[k * size + j]);
																							            }
																		            m_out[i * size + j] = (m_in[i * size + j] - sum) / m_out[i * size + i];
																					        }
											    }
}



void standard (float *A, float  *B, float *C, int size)
{
	    int i, j, k;

		    for (i = 0; i < size; i++)
				        for (j = 0; j < size; j++)
							            for (k = 0; k < size; k++)

										{

											                C[i * size + j] += A[k * size + i] * B[k * size + j]; 
															if (i==j)C[i*size+j]+= 0.001;
										}
}

void init(float *v, int n)
{
	    int i;
		    srand(time(NULL));
			    for (i = 0; i < n; i++)
					        v[i] = rand() / (float(RAND_MAX) + 1)-1; 
}



void loadMatrix(float * matrix, char *s, int size)
{
	fstream f;
	int i = 0;
	f.open(s, ifstream::in);
	while (f.good()) {
		f >> matrix[i];
		i++;
	}
	f.close();
}

void saveMatrix(float * matrix, char *s, int size)
{
	fstream f;
	f.open(s, ifstream::out);
	for(int i = 0; i < size; i++) {
		for(int j = 0; j < size; j++) {
			f << matrix[i * size + j] << " ";
		}
		f << endl;
	}
	f.close();
}


__global__ void gpu_dpotrf(float *m, int size, int p)
{
	int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float a[16][16+1];
    a[ty][tx] = m[(ty + 16 * p) * size + tx + 16 * p];

    __syncthreads();

    float fac;

	#pragma unroll 16
    for (int k = 0; k < 16; k++)
    {
		__syncthreads();
		fac = rsqrt(a[k][k]);
		__syncthreads();
		if ((ty == k) && (tx >= k)) 
	    	a[tx][ty] = (a[tx][ty]) * fac;
	
		__syncthreads();

		if ((ty >= tx) && (tx > k)) 
	    	a[ty][tx]=a[ty][tx] - a[tx][k]*a[ty][k]; 
	

    }

    __syncthreads();

    if (ty>=tx) 
	m[(tx+16*p)*size+ty+16*p]=a[ty][tx];
    


}

__global__ void gpu_inv_l(float *u, int size, int p)
{
	int i, j;

	int tid = threadIdx.x;
	int bx=blockIdx.x+1;
	__shared__ float b[16][16];
	for(i=0;i<16;i++)
		b[i][tid]=u[(i+p*16)*size+tid+(bx+p)*16];
	b[0][tid] = b[0][tid] / 
		u[(0 + p * 16) * size + (0 + 16 * p)];
	for (i = 1; i < 16; i++){
		for (j = 0; j < i; j++){
			b[i][tid] = b[i][tid] - 
				u[(j + p * 16) * size + (i + p * 16)] *
				b[j][tid];
		}
		b[i][tid] = b[i][tid] / u[(i + p * 16) * size + 
							(i + 16 *p)];

	}
	for(i=0;i<16;i++)
		u[(i+p*16)*size+tid+(bx+p)*16]=b[i][tid];
}

__global__ void gpu_mm_a(float *m, int size, int p, int it)
{
	__shared__ float s_a[16][16];
	__shared__ float s_b[16][16];
	__shared__ float s_c[16][16];
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx_g = blockIdx.x;
	int by_g = blockIdx.y;
	int i;

	int bx=0, by=0, o = 0, e = -1;

		if(!(it % 2)){
			o = 1;
			e = -2;
		}

		int pi, n = it;

		pi = n - 1 - by_g;
		if(bx_g <= pi){
			by=by_g;
			bx=by_g+bx_g;
		}
		else{
			by=n-(by_g+o);
			bx=(n-by_g)+bx_g-pi+e;
		}

	s_a[ty][tx] = m[(ty + p * 16) * size + tx + (p + 1) * 16 + by * 16];
	s_b[ty][tx] = m[(ty + p * 16) * size + tx + (p + 1) * 16 + bx * 16];
	s_c[ty][tx] = 0;

	__syncthreads();

	#pragma unroll 16
	for (i = 0; i < 16; i++)
	{
		s_c[ty][tx] += s_a[i][ty] * s_b[i][tx];
	}

	m[(ty + (p + 1 + by) * 16) * size + tx + (p + 1 + bx) * 16] -= s_c[ty][tx];
}


__global__ void gpu_mm_r(float *a, float *b, int size, int p)
{
	__shared__ float s_a[16][16];
	__shared__ float s_b[16][16];
	__shared__ float s_c[16][16];
	
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int stride = blockIdx.x + 1;
	int i;

	s_a[ty][tx] = a[ty * 16 + tx];
	s_b[ty][tx] = b[(ty + p * 16) * size + tx + 16 * (stride + p)];
	s_c[ty][tx] = 0;
	__syncthreads();

	#pragma unroll 16
	for (i = 0; i < 16; i++)
	{
		s_c[ty][tx] += s_a[ty][i] * s_b[i][tx];
	}
	b[(ty + p * 16) * size + tx + 16 * (stride + p)] = s_c[ty][tx];
}

void init_eye(float *v, int n)
{
	int i;
	for (i = 0; i < n; i++)
		v[i * n + i] = 1.;
}

int main(int argc, char *argv[])
{
	int size = 1024;

	unsigned int timer2 = 0, t = 0, t2 = 0, ta = 0;

	float *m_in, *m_out, *device_m, *device_m_out, *eye, *device_eye, *v, *cpu_rez;
	m_in = new float[size * size];
	m_out = new float[size * size];
	cpu_rez = new float[size * size];
	v = new float[size * size];
	eye = new float[16 * 16];

	memset(m_in, 0, size * size * sizeof(float));
	memset(m_out, 0, size * size * sizeof(float));
	memset(cpu_rez, 0, size * size * sizeof(float));
	memset(eye, 0, 16 * 16 * sizeof(float));

	init_eye(eye, 16);


	int deviceOrdinal = 0;
	hipSetDevice(deviceOrdinal);
	hipDeviceProp_t device_properties;
	hipGetDeviceProperties(&device_properties, deviceOrdinal);
	printf("%s\n\n", device_properties.name);

	printf("Generiranje matrice ");

	CUT_SAFE_CALL(cutCreateTimer(&t));
	CUT_SAFE_CALL(cutStartTimer(t));
	
	//loadMatrix(m_in, "matrice/po32.mat", size);
	init(v, size * size);
	standard(v, v, m_in, size);

	CUT_SAFE_CALL(cutStopTimer(t));

	printf("%f\n", cutGetTimerValue(t));

	printf("CPU racuna ");

	CUT_SAFE_CALL(cutCreateTimer(&ta));
	CUT_SAFE_CALL(cutStartTimer(ta));
	
	//loadMatrix(m_in, "matrice/po32.mat", size);

	cpu_dpotrf(m_in, cpu_rez, size);
	CUT_SAFE_CALL(cutStopTimer(ta));

	printf("%f\n", cutGetTimerValue(ta));
	// GPU //
	int n = size;
	saveMatrix(m_in, "m.mat", n);
	saveMatrix(cpu_rez, "cpu_rez.mat", n);
	dim3 blokovaPoGridu, thredovaPoBloku;
	
	thredovaPoBloku.x = 16;
	thredovaPoBloku.y = 16;

	hipMalloc((void **) &device_m, n * n * sizeof(float));
	hipMalloc((void **) &device_m_out, n * n * sizeof(float));
	hipMalloc((void **) &device_eye, 16 * 16 * sizeof(float));

	hipMemset(device_m_out, 0, n * n *sizeof(float));
	

	printf("Kopiranje matrice na GPU: ");

	CUT_SAFE_CALL(cutCreateTimer(&t2));
	CUT_SAFE_CALL(cutStartTimer(t2));

	hipMemcpy( device_m, 
				m_in, 
				n * n * sizeof(float), 
				hipMemcpyHostToDevice );

/*	hipMemcpy( device_eye, 
				eye, 
				16 * 16 * sizeof(float), 
				hipMemcpyHostToDevice );*/

	CUT_SAFE_CALL(cutStopTimer(t2));

	printf("%f\n", cutGetTimerValue(t2));

	printf("GPU racuna: ");

	hipDeviceSynchronize();

	CUT_SAFE_CALL(cutCreateTimer(&timer2));
	CUT_SAFE_CALL(cutStartTimer(timer2));

	int i;
	int it = n / 16 - 1;
	gpu_dpotrf<<< 1, 
				  thredovaPoBloku, 4*16*16*sizeof(float)
				   >>>
				  ( device_m, size, 0 );
	hipDeviceSynchronize();

	for (i = 0; i < n / 16 - 1; i++) {
		blokovaPoGridu.x = it;
		blokovaPoGridu.y = it;
		gpu_inv_l<<<it, 16, 4*16*16*sizeof(float)>>>(device_m, size, i);
	hipDeviceSynchronize();
//		gpu_mm_r<<<it, thredovaPoBloku>>>
//			(device_eye, device_m, size, i);
		if(it % 2){
			blokovaPoGridu.y = (it+1)/2;
			blokovaPoGridu.x = it;
		}
		else{
			blokovaPoGridu.y = it/2;
			blokovaPoGridu.x = it+1;
		}
	//	blokovaPoGridu.x = 1;
	//		blokovaPoGridu.y = 1;
	//blokovaPoGridu.y = it;
	//blokovaPoGridu.x = it;
		gpu_mm_a<<<blokovaPoGridu, thredovaPoBloku, 4*16*16*sizeof(float)>>>
		(device_m, size, i, it);
	hipDeviceSynchronize();
		gpu_dpotrf<<<1, thredovaPoBloku, 4*16*16*sizeof(float)>>>
		(device_m, size, i + 1);
	hipDeviceSynchronize();
		it--;
		
	}
	/*gpu_dpotrf<<< 1, thredovaPoBloku>>> ( device_m, size, 0 );


	hipMemcpy( device_eye, eye, 16 * 16 * sizeof(float), hipMemcpyHostToDevice );
	gpu_inv_l<<<1, 16>>>(device_m, device_eye, size, 0);
	gpu_mm_r<<<it, thredovaPoBloku>>> (device_eye, device_m, size, 0);
	blokovaPoGridu.y = it;
	blokovaPoGridu.x = it;
	gpu_mm_a<<<blokovaPoGridu, thredovaPoBloku>>>(device_m, size, 0, it);
	gpu_dpotrf<<<1, thredovaPoBloku>>>	(device_m, size, 0 + 1);
	it--;
	hipMemcpy( device_eye, eye, 16 * 16 * sizeof(float), hipMemcpyHostToDevice );
	gpu_inv_l<<<1, 16>>>(device_m, device_eye, size, 1);
	gpu_mm_r<<<it, thredovaPoBloku>>> (device_eye, device_m, size, 1);
	blokovaPoGridu.y = it;
	blokovaPoGridu.x = it;
	gpu_mm_a<<<blokovaPoGridu, thredovaPoBloku, 3 * 16 * 16 * sizeof(float)>>>(device_m, size, 1, it);
	gpu_dpotrf<<<1, thredovaPoBloku, 16 * 16 * sizeof(float)>>>	(device_m, size, 1 + 1);
	it--;
	hipMemcpy( device_eye, eye, 16 * 16 * sizeof(float), hipMemcpyHostToDevice );
	gpu_inv_l<<<1, 16>>>(device_m, device_eye, size, 2);
	gpu_mm_r<<<it, thredovaPoBloku, 3 * 16 * 16 * sizeof(float)>>> (device_eye, device_m, size, 2);
	blokovaPoGridu.y = it;
	blokovaPoGridu.x = it;
	gpu_mm_a<<<blokovaPoGridu, thredovaPoBloku, 3 * 16 * 16 * sizeof(float)>>>(device_m, size, 2, it);
	gpu_dpotrf<<<1, thredovaPoBloku, 16 * 16 * sizeof(float)>>>	(device_m, size, 2 + 1);*/


	hipDeviceSynchronize();

	CUT_SAFE_CALL(cutStopTimer(timer2));

	printf("%f\n", cutGetTimerValue(timer2));

	
	hipMemcpy(m_out, device_m, 
			n * n * sizeof(float), hipMemcpyDeviceToHost);
	printf("CPU %.20lf\n", cpu_rez[(size-1)*size+size-1]);
	printf("GPU %.20lf\n", m_out[(size-1)*size+size-1]);

	saveMatrix(m_out, "rez.mat", n);


	free(m_in);
	free(m_out);
	free(eye);
	hipFree(device_m);
	hipFree(device_m_out);
	hipFree(device_eye);

	return 0;
}
